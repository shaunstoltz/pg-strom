#include "hip/hip_runtime.h"
/*
 * libgpuscan.cu
 *
 * GPU implementation of GpuScan
 * ----
 * Copyright 2011-2020 (C) KaiGai Kohei <kaigai@kaigai.gr.jp>
 * Copyright 2014-2020 (C) The PG-Strom Development Team
 *
 * This program is free software; you can redistribute it and/or modify
 * it under the terms of the GNU General Public License version 2 as
 * published by the Free Software Foundation.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 */
#include "cuda_common.h"
#include "cuda_gpuscan.h"

/*
 * gpuscan_main_row - GpuScan logic for KDS_FORMAT_ROW
 */
DEVICE_FUNCTION(void)
gpuscan_main_row(kern_context *kcxt,
				 kern_gpuscan *kgpuscan,
				 kern_data_store *kds_src,
				 kern_data_store *kds_dst,
				 bool has_device_projection)
{
	gpuscanSuspendContext *my_suspend
		= KERN_GPUSCAN_SUSPEND_CONTEXT(kgpuscan, get_group_id());
	gpuscanResultIndex *gs_results	__attribute__((unused))
		= KERN_GPUSCAN_RESULT_INDEX(kgpuscan);
	cl_uint		dst_ncols = (kds_dst ? kds_dst->ncols : kds_src->ncols);
	cl_uint		part_index = 0;
	cl_uint		src_index;
	cl_uint		src_base;
	cl_uint		nitems_offset;
	cl_uint		usage_offset	__attribute__((unused));
	cl_uint		total_nitems_in = 0;	/* stat */
	cl_uint		total_nitems_out = 0;	/* stat */
	cl_uint		total_extra_size = 0;	/* stat */
	__shared__ cl_uint	nitems_base;
	__shared__ cl_ulong	usage_base	__attribute__((unused));

	assert(kds_src->format == KDS_FORMAT_ROW);
	assert(!kds_dst || kds_dst->format == KDS_FORMAT_ROW);
	/* quick bailout if any error happen on the prior kernel */
	if (__syncthreads_count(kgpuscan->kerror.errcode) != 0)
		return;
	/* resume kernel from the point where suspended, if any */
	if (kgpuscan->resume_context)
	{
		assert(my_suspend != NULL);
		part_index = my_suspend->part_index;
	}

	for (src_base = get_global_base() + part_index * get_global_size();
		 src_base < kds_src->nitems;
		 src_base += get_global_size(), part_index++)
	{
		kern_tupitem   *tupitem;
		cl_bool			rc;
		cl_uint			nvalids;
		cl_uint			required	__attribute__((unused));
		cl_uint			extra_sz = 0;
		cl_uint			suspend_kernel	__attribute__((unused)) = 0;
		cl_char		   *tup_dclass = NULL;
		Datum		   *tup_values = NULL;

		/* rewind the varlena buffer */
		kcxt->vlpos = kcxt->vlbuf;
		/* Evalidation of the rows by WHERE-clause */
		src_index = src_base + get_local_id();
		if (src_index < kds_src->nitems)
		{
			tupitem = KERN_DATA_STORE_TUPITEM(kds_src, src_index);
			rc = gpuscan_quals_eval(kcxt, kds_src,
									&tupitem->t_self,
									&tupitem->htup);
		}
		else
		{
			tupitem = NULL;
			rc = false;
		}
		/* bailout if any error */
		if (__syncthreads_count(kcxt->errcode) > 0)
			goto out_nostat;

		/* how many rows servived WHERE-clause evaluation? */
		nitems_offset = pgstromStairlikeBinaryCount(tupitem && rc, &nvalids);
		if (nvalids == 0)
			goto skip;

		if (has_device_projection)
		{
			/* extract the source tuple to the private slot, if any */
			if (tupitem && rc)
			{
				kcxt->vlpos = kcxt->vlbuf;	/* rewind */
				tup_dclass = (cl_char *)
					kern_context_alloc(kcxt, sizeof(cl_char) * dst_ncols);
				tup_values = (Datum *)
					kern_context_alloc(kcxt, sizeof(Datum) * dst_ncols);

				if (!tup_dclass || !tup_values)
				{
					required = 0;
					STROM_CPU_FALLBACK(kcxt, ERRCODE_OUT_OF_MEMORY,
									   "out of memory");
				}
				else
				{
					gpuscan_projection_tuple(kcxt,
											 kds_src,
											 &tupitem->htup,
											 &tupitem->t_self,
											 tup_dclass,
											 tup_values);
					required = MAXALIGN(offsetof(kern_tupitem, htup) +
										compute_heaptuple_size(kcxt,
															   kds_dst,
															   tup_dclass,
															   tup_values));
				}
			}
			else
				required = 0;
			/* bailout if any error */
			if (__syncthreads_count(kcxt->errcode) > 0)
				goto out_nostat;

			usage_offset = pgstromStairlikeSum(required, &extra_sz);
			if (get_local_id() == 0)
			{
				union {
					struct {
						cl_uint	nitems;
						cl_uint	usage;
					} i;
					cl_ulong	v64;
				} oldval, curval, newval;

				curval.i.nitems	= kds_dst->nitems;
				curval.i.usage	= kds_dst->usage;
				do {
					newval = oldval = curval;
					newval.i.nitems += nvalids;
					newval.i.usage  += __kds_packed(extra_sz);

					if (KERN_DATA_STORE_HEAD_LENGTH(kds_dst) +
						STROMALIGN(sizeof(cl_uint) * newval.i.nitems) +
						__kds_unpack(newval.i.usage) > kds_dst->length)
					{
						atomicAdd(&kgpuscan->suspend_count, 1);
						suspend_kernel = 1;
						break;
					}
				} while ((curval.v64 = atomicCAS((cl_ulong *)&kds_dst->nitems,
												 oldval.v64,
												 newval.v64)) != oldval.v64);
				nitems_base = oldval.i.nitems;
				usage_base  = __kds_unpack(oldval.i.usage);
			}
			if (__syncthreads_count(suspend_kernel) > 0)
				break;

			/* store the result heap-tuple on destination buffer */
			if (tupitem && rc)
			{
				cl_uint	   *tup_index = KERN_DATA_STORE_ROWINDEX(kds_dst);
				size_t		pos;

				pos = kds_dst->length - (usage_base + usage_offset + required);
				tup_index[nitems_base + nitems_offset] = __kds_packed(pos);
				form_kern_heaptuple(kcxt,
									(kern_tupitem *)((char *)kds_dst + pos),
									kds_dst,
									&tupitem->t_self,
									&tupitem->htup,
									tup_dclass,
									tup_values);
			}
		}
		else
		{
			if (get_local_id() == 0)
				nitems_base = atomicAdd(&gs_results->nitems, nvalids);
			__syncthreads();
			if (tupitem && rc)
			{
				assert(nitems_base + nitems_offset < kds_src->nrooms);
				gs_results->results[nitems_base + nitems_offset]
					= __kds_packed((char *)&tupitem->htup -
								   (char *)kds_src);
			}
		}
	skip:
		/* update statistics */
		if (get_local_id() == 0)
		{
			total_nitems_in  += Min(kds_src->nitems - src_base,
									get_local_size());
			total_nitems_out += nvalids;
			total_extra_size += extra_sz;
		}
	}
	/* write back statistics and error code */
	if (get_local_id() == 0)
	{
		atomicAdd(&kgpuscan->nitems_in,  total_nitems_in);
		atomicAdd(&kgpuscan->nitems_out, total_nitems_out);
		atomicAdd(&kgpuscan->extra_size, total_extra_size);
	}
out_nostat:
	/* suspend the current position (even if normal exit) */
	if (my_suspend && get_local_id() == 0)
	{
		my_suspend->part_index = part_index;
		my_suspend->line_index = 0;
	}
}

/*
 * gpuscan_main_block - GpuScan logic for KDS_FORMAT_BLOCK
 */
DEVICE_FUNCTION(void)
gpuscan_main_block(kern_context *kcxt,
				   kern_gpuscan *kgpuscan,
				   kern_data_store *kds_src,
				   kern_data_store *kds_dst,
				   bool has_device_projection)
{
	gpuscanSuspendContext *my_suspend
		= KERN_GPUSCAN_SUSPEND_CONTEXT(kgpuscan, get_group_id());
	cl_uint		src_nitems = kds_src->nitems;
	cl_uint		dst_ncols = kds_dst->ncols;
	cl_uint		part_sz;
	cl_uint		n_parts;
	cl_uint		nitems_offset;
	cl_uint		usage_offset;
	cl_uint		window_sz;
	cl_uint		part_base;
	cl_uint		part_index = 0;
	cl_uint		line_index = 0;
	cl_uint		total_nitems_in = 0;	/* stat */
	cl_uint		total_nitems_out = 0;	/* stat */
	cl_uint		total_extra_size = 0;	/* stat */
	cl_bool		thread_is_valid = false;
	__shared__ cl_uint	nitems_base;
	__shared__ cl_ulong	usage_base;

	assert(kds_src->format == KDS_FORMAT_BLOCK);
	assert(kds_dst->format == KDS_FORMAT_ROW);
	/* quick bailout if any error happen on the prior kernel */
	if (__syncthreads_count(kgpuscan->kerror.errcode) != 0)
		return;

	part_sz = KERN_DATA_STORE_PARTSZ(kds_src);
	n_parts = get_local_size() / part_sz;
	if (get_global_id() == 0)
		kgpuscan->part_sz = part_sz;
	if (get_local_id() < part_sz * n_parts)
		thread_is_valid = true;
	window_sz = n_parts * get_num_groups();

	/* resume kernel from the point where suspended, if any */
	if (kgpuscan->resume_context)
	{
		part_index = my_suspend->part_index;
		line_index = my_suspend->line_index;
	}
	__syncthreads();

	for (;;)
	{
		cl_uint		part_id;
		cl_uint		line_no;
		cl_uint		n_lines = 0;
		cl_uint		nvalids;
		cl_uint		nitems_real;

		part_base = part_index * window_sz + get_group_id() * n_parts;
		if (part_base >= kds_src->nitems)
			break;
		part_id = get_local_id() / part_sz + part_base;
		line_no = get_local_id() % part_sz + line_index * part_sz;

		do {
			HeapTupleHeaderData *htup = NULL;
			ItemPointerData t_self;
			PageHeaderData *pg_page;
			BlockNumber	block_nr;
			cl_ushort	t_len	__attribute__((unused));
			cl_uint		required;
			cl_uint		extra_sz = 0;
			cl_uint		suspend_kernel = 0;
			cl_bool		rc;
			cl_char	   *tup_dclass = NULL;
			Datum	   *tup_values = NULL;

			/* rewind the varlena buffer */
			kcxt->vlpos = kcxt->vlbuf;

			/* identify the block */
			if (thread_is_valid && part_id < src_nitems)
			{
				pg_page = KERN_DATA_STORE_BLOCK_PGPAGE(kds_src, part_id);
				n_lines = PageGetMaxOffsetNumber(pg_page);
				block_nr = KERN_DATA_STORE_BLOCK_BLCKNR(kds_src, part_id);
				t_self.ip_blkid.bi_hi = block_nr >> 16;
				t_self.ip_blkid.bi_lo = block_nr & 0xffff;
				t_self.ip_posid = line_no + 1;

				if (line_no < n_lines)
				{
					ItemIdData *lpp = PageGetItemId(pg_page, line_no+1);
					if (ItemIdIsNormal(lpp))
						htup = PageGetItem(pg_page, lpp);
					t_len = ItemIdGetLength(lpp);
				}
			}

			/* evaluation of the qualifiers */
			if (htup)
				rc = gpuscan_quals_eval(kcxt, kds_src,
										&t_self,
										htup);
			else
				rc = false;
			/* bailout if any error */
			if (__syncthreads_count(kcxt->errcode) > 0)
				goto out_nostat;

			/* how many rows servived WHERE-clause evaluations? */
			nitems_offset = pgstromStairlikeBinaryCount(htup && rc, &nvalids);
			if (nvalids == 0)
				goto skip;

			/* store the result heap-tuple to destination buffer */
			if (htup && rc)
			{
				if (has_device_projection)
				{
					tup_dclass = (cl_char *)
						kern_context_alloc(kcxt, sizeof(cl_char) * dst_ncols);
					tup_values = (Datum *)
						kern_context_alloc(kcxt, sizeof(Datum) * dst_ncols);

					if (!tup_dclass || !tup_values)
					{
						required = 0;
						STROM_EREPORT(kcxt, ERRCODE_OUT_OF_MEMORY,
									  "out of memory");
					}
					else
					{
						gpuscan_projection_tuple(kcxt,
												 kds_src,
												 htup,
												 &t_self,
												 tup_dclass,
												 tup_values);
						required = MAXALIGN(offsetof(kern_tupitem, htup) +
											compute_heaptuple_size(kcxt,
																   kds_dst,
																   tup_dclass,
																   tup_values));
					}
				}
				else
				{
					/* no projection; just write the source tuple as is */
					required = MAXALIGN(offsetof(kern_tupitem, htup) + t_len);
				}
			}
			else
				required = 0;
			/* bailout if any error */
			if (__syncthreads_count(kcxt->errcode) > 0)
				goto out;

			usage_offset = pgstromStairlikeSum(required, &extra_sz);
			if (get_local_id() == 0)
			{
				union {
					struct {
						cl_uint	nitems;
						cl_uint	usage;
					} i;
					cl_ulong	v64;
				} oldval, curval, newval;

				curval.i.nitems = kds_dst->nitems;
				curval.i.usage  = kds_dst->usage;
				do {
					newval = oldval = curval;
					newval.i.nitems += nvalids;
					newval.i.usage  += __kds_packed(extra_sz);

					if (KERN_DATA_STORE_HEAD_LENGTH(kds_dst) +
						STROMALIGN(sizeof(cl_uint) * newval.i.nitems) +
						__kds_unpack(newval.i.usage) > kds_dst->length)
					{
						atomicAdd(&kgpuscan->suspend_count, 1);
						suspend_kernel = 1;
						break;
					}
				} while ((curval.v64 = atomicCAS((cl_ulong *)&kds_dst->nitems,
												 oldval.v64,
												 newval.v64)) != oldval.v64);
				nitems_base = oldval.i.nitems;
				usage_base  = __kds_unpack(oldval.i.usage);
			}
			if (__syncthreads_count(suspend_kernel) > 0)
				goto out;

			/* store the result heap tuple */
			if (htup && rc)
			{
				cl_uint	   *tup_index = KERN_DATA_STORE_ROWINDEX(kds_dst);
				size_t		pos = (kds_dst->length
								   - (usage_base + usage_offset + required));
				if (has_device_projection)
				{
					form_kern_heaptuple(kcxt,
										(kern_tupitem *)((char *)kds_dst + pos),
										kds_dst,
										&t_self,
										htup,
										tup_dclass,
										tup_values);
				}
				else
				{
					kern_tupitem *tupitem;

					tupitem = (kern_tupitem *)((char *)kds_dst + pos);
					tupitem->t_len = t_len;
					tupitem->t_self = t_self;
					memcpy(&tupitem->htup, htup, t_len);
				}
				tup_index[nitems_base + nitems_offset] = __kds_packed(pos);
			}
		skip:
			/* update statistics */
			pgstromStairlikeBinaryCount(htup != NULL, &nitems_real);
			if (get_local_id() == 0)
			{
				total_nitems_in		+= nitems_real;
				total_nitems_out	+= nvalids;
				total_extra_size	+= extra_sz;
			}

			/*
			 * Move to the next window of the line items, if any.
			 * If no threads in CUDA block wants to continue, exit the loop.
			 */
			line_index++;
			line_no += part_sz;
		} while (__syncthreads_count(thread_is_valid &&
									 line_no < n_lines) > 0);
		/* move to the next window */
		part_index++;
		line_index = 0;
	}
out:
	/* update statistics */
	if (get_local_id() == 0)
	{
		atomicAdd(&kgpuscan->nitems_in,  total_nitems_in);
		atomicAdd(&kgpuscan->nitems_out, total_nitems_out);
		atomicAdd(&kgpuscan->extra_size, total_extra_size);
	}
out_nostat:
	if (get_local_id() == 0)
	{
		my_suspend->part_index = part_index;
		my_suspend->line_index = line_index;
	}
}

/*
 * gpuscan_main_arrow - GpuScan logic for KDS_FORMAT_ARROW
 */
DEVICE_FUNCTION(void)
gpuscan_main_arrow(kern_context *kcxt,
				   kern_gpuscan *kgpuscan,
				   kern_data_store *kds_src,
				   kern_data_store *kds_dst,
				   bool has_device_projection)
{
	gpuscanSuspendContext *my_suspend
		= KERN_GPUSCAN_SUSPEND_CONTEXT(kgpuscan, get_group_id());
	cl_uint		part_index = 0;
	cl_uint		dst_ncols = kds_dst->ncols;
	cl_uint		src_base;
	cl_uint		src_index;
	cl_uint		nitems_offset;
	cl_uint		usage_offset	__attribute__((unused));
	cl_uint		total_nitems_in = 0;	/* stat */
	cl_uint		total_nitems_out = 0;	/* stat */
	cl_uint		total_extra_size = 0;	/* stat */
	__shared__ cl_uint	nitems_base;
	__shared__ cl_ulong	usage_base	__attribute__((unused));

	assert(kds_src->format == KDS_FORMAT_ARROW);
	assert(!kds_dst || kds_dst->format == KDS_FORMAT_ROW);
	/* quick bailout if any error happen on the prior kernel */
	if (__syncthreads_count(kgpuscan->kerror.errcode) != 0)
		return;
	/* resume kernel from the point where suspended, if any */
	if (kgpuscan->resume_context)
	{
		assert(my_suspend != NULL);
		part_index = my_suspend->part_index;
	}

	for (src_base = get_global_base() + part_index * get_global_size();
		 src_base < kds_src->nitems;
		 src_base += get_global_size(), part_index++)
	{
		kern_tupitem   *tupitem		__attribute__((unused));
		cl_bool			rc;
		cl_uint			nvalids;
		cl_uint			required	__attribute__((unused));
		cl_uint			extra_sz = 0;
		cl_uint			suspend_kernel = 0;
		cl_char		   *tup_dclass = NULL;
		Datum		   *tup_values = NULL;

		/* rewind the varlena buffer */
		kcxt->vlpos = kcxt->vlbuf;

		/* Evalidation of the rows by WHERE-clause */
		src_index = src_base + get_local_id();
		if (src_index < kds_src->nitems)
			rc = gpuscan_quals_eval_arrow(kcxt, kds_src, src_index);
		else
			rc = false;
		/* bailout if any error */
		if (__syncthreads_count(kcxt->errcode) > 0)
			goto out_nostat;

		/* how many rows servived WHERE-clause evaluation? */
		nitems_offset = pgstromStairlikeBinaryCount(rc, &nvalids);
		if (nvalids == 0)
			goto skip;

		/*
		 * OK, extract the source columns to form a result row
		 */
		if (rc)
		{
			kcxt->vlpos = kcxt->vlbuf;	/* rewind */
			tup_dclass = (cl_char *)
				kern_context_alloc(kcxt, sizeof(cl_char) * dst_ncols);
			tup_values = (Datum *)
				kern_context_alloc(kcxt, sizeof(Datum) * dst_ncols);

			if (!tup_dclass || !tup_values)
			{
				required = 0;
				STROM_EREPORT(kcxt, ERRCODE_OUT_OF_MEMORY,
							  "out of memory");
			}
			else
			{
				gpuscan_projection_arrow(kcxt,
										 kds_src,
										 src_index,
										 tup_dclass,
										 tup_values);
				required = MAXALIGN(offsetof(kern_tupitem, htup) +
									compute_heaptuple_size(kcxt,
														   kds_dst,
														   tup_dclass,
														   tup_values));
			}
		}
		else
			required = 0;

		usage_offset = pgstromStairlikeSum(required, &extra_sz);
		if (get_local_id() == 0)
		{
			union {
				struct {
					cl_uint	nitems;
					cl_uint	usage;
				} i;
				cl_ulong	v64;
			} oldval, curval, newval;

			curval.i.nitems = kds_dst->nitems;
			curval.i.usage  = kds_dst->usage;
			do {
				newval = oldval = curval;
				newval.i.nitems += nvalids;
				newval.i.usage  += __kds_packed(extra_sz);

				if (KERN_DATA_STORE_HEAD_LENGTH(kds_dst) +
					STROMALIGN(sizeof(cl_uint) * newval.i.nitems) +
					__kds_unpack(newval.i.usage) > kds_dst->length)
				{
					atomicAdd(&kgpuscan->suspend_count, 1);
					suspend_kernel = 1;
					break;
				}
			} while ((curval.v64 = atomicCAS((cl_ulong *)&kds_dst->nitems,
											 oldval.v64,
											 newval.v64)) != oldval.v64);
			nitems_base = oldval.i.nitems;
			usage_base  = __kds_unpack(oldval.i.usage);
		}
		if (__syncthreads_count(suspend_kernel) > 0)
			break;

		/* store the result heap-tuple on destination buffer */
		if (required > 0)
		{
			cl_uint	   *tup_index = KERN_DATA_STORE_ROWINDEX(kds_dst);
			cl_uint		pos;

			pos = kds_dst->length - (usage_base + usage_offset + required);
			tup_index[nitems_base + nitems_offset] = __kds_packed(pos);
			form_kern_heaptuple(kcxt,
								(kern_tupitem *)((char *)kds_dst + pos),
								kds_dst,
								NULL,	/* ItemPointerData */
								NULL,	/* HeapTupleHeaderData */
								tup_dclass,
								tup_values);
		}
		/* bailout if any error */
		if (__syncthreads_count(kcxt->errcode) > 0)
			break;
	skip:
		/* update statistics */
		if (get_local_id() == 0)
		{
			total_nitems_in  += Min(kds_src->nitems - src_base,
									get_local_size());
			total_nitems_out += nvalids;
			total_extra_size += extra_sz;
		}
	}
	/* write back statistics and error code */
	if (get_local_id() == 0)
	{
		atomicAdd(&kgpuscan->nitems_in,  total_nitems_in);
		atomicAdd(&kgpuscan->nitems_out, total_nitems_out);
		atomicAdd(&kgpuscan->extra_size, total_extra_size);
	}
out_nostat:
	/* suspend the current position (even if normal exit) */
	if (my_suspend && get_local_id() == 0)
	{
		my_suspend->part_index = part_index;
		my_suspend->line_index = 0;
	}
}
