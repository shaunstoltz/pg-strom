#include "hip/hip_runtime.h"
/*
 * cuda_primitive.cu
 *
 * Collection of primitive functions for CUDA GPU devices
 * --
 * Copyright 2011-2020 (C) KaiGai Kohei <kaigai@kaigai.gr.jp>
 * Copyright 2014-2020 (C) The PG-Strom Development Team
 *
 * This program is free software; you can redistribute it and/or modify
 * it under the terms of the GNU General Public License version 2 as
 * published by the Free Software Foundation.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 */
#include "cuda_common.h"
#include "cuda_primitive.h"

/*
 * Utility macros
 */
#define CHECKFLOATVAL(kerror, result, inf_is_valid, zero_is_valid)	\
	do {															\
		if (isinf((result).value) && !(inf_is_valid))				\
		{															\
			(result).isnull = true;									\
			STROM_EREPORT(kcxt, ERRCODE_NUMERIC_VALUE_OUT_OF_RANGE,	\
						  "value out of range: overflow");			\
		}															\
		if ((result).value == 0.0 && !(zero_is_valid))				\
		{															\
			(result).isnull = true;									\
			STROM_EREPORT(kcxt, ERRCODE_NUMERIC_VALUE_OUT_OF_RANGE,	\
						  "value out of range: underflow");			\
		}															\
	} while(0)

#ifndef SAMESIGN
#define SAMESIGN(a,b)	(((a) < 0) == ((b) < 0))
#endif

/*
 * Functions for addition operator on basic data types
 */
#define BASIC_INT_ADDFUNC_TEMPLATE(name,r_type,x_type,y_type)		\
	DEVICE_FUNCTION(pg_##r_type##_t)								\
	pgfn_##name(kern_context *kcxt,									\
				pg_##x_type##_t arg1, pg_##y_type##_t arg2)			\
	{																\
		pg_##r_type##_t	result;										\
																	\
		result.isnull = arg1.isnull | arg2.isnull;					\
		if (!result.isnull)											\
		{															\
			result.value = arg1.value + arg2.value;					\
			if (SAMESIGN(arg1.value, arg2.value) &&					\
				!SAMESIGN(result.value, arg1.value))				\
			{														\
				result.isnull = true;								\
				STROM_EREPORT(kcxt, ERRCODE_NUMERIC_VALUE_OUT_OF_RANGE,\
							  "integer out of range");				\
			}														\
		}															\
		return result;												\
	}

#define BASIC_FLOAT_ADDFUNC_TEMPLATE(name,r_type,x_type,y_type,cast_t) \
	DEVICE_FUNCTION(pg_##r_type##_t)								\
	pgfn_##name(kern_context *kcxt,									\
				pg_##x_type##_t arg1, pg_##y_type##_t arg2)         \
    {																\
		pg_##r_type##_t	result;										\
																	\
		result.isnull = arg1.isnull | arg2.isnull;					\
		if (!result.isnull)											\
		{															\
			result.value = (cast_t)arg1.value + (cast_t)arg2.value;	\
			CHECKFLOATVAL(&kcxt->e, result,							\
						  isinf((cast_t)arg1.value) ||				\
						  isinf((cast_t)arg2.value), true);			\
		}                                                           \
		return result;												\
	}
BASIC_INT_ADDFUNC_TEMPLATE(int1pl, int1,int1,int1)
BASIC_INT_ADDFUNC_TEMPLATE(int12pl,int2,int1,int2)
BASIC_INT_ADDFUNC_TEMPLATE(int14pl,int4,int1,int4)
BASIC_INT_ADDFUNC_TEMPLATE(int18pl,int8,int1,int8)

BASIC_INT_ADDFUNC_TEMPLATE(int21pl,int2,int2,int1)
BASIC_INT_ADDFUNC_TEMPLATE(int2pl, int2,int2,int2)
BASIC_INT_ADDFUNC_TEMPLATE(int24pl,int4,int2,int4)
BASIC_INT_ADDFUNC_TEMPLATE(int28pl,int8,int2,int8)

BASIC_INT_ADDFUNC_TEMPLATE(int41pl,int4,int4,int1)
BASIC_INT_ADDFUNC_TEMPLATE(int42pl,int4,int4,int2)
BASIC_INT_ADDFUNC_TEMPLATE(int4pl, int4,int4,int4)
BASIC_INT_ADDFUNC_TEMPLATE(int48pl,int8,int4,int8)

BASIC_INT_ADDFUNC_TEMPLATE(int81pl,int8,int8,int1)
BASIC_INT_ADDFUNC_TEMPLATE(int82pl,int8,int8,int2)
BASIC_INT_ADDFUNC_TEMPLATE(int84pl,int8,int8,int4)
BASIC_INT_ADDFUNC_TEMPLATE(int8pl, int8,int8,int8)

BASIC_FLOAT_ADDFUNC_TEMPLATE(float2pl, float4, float2, float2, cl_float)
BASIC_FLOAT_ADDFUNC_TEMPLATE(float24pl,float4, float2, float4, cl_float)
BASIC_FLOAT_ADDFUNC_TEMPLATE(float28pl,float8, float2, float8, cl_double)
BASIC_FLOAT_ADDFUNC_TEMPLATE(float42pl,float4, float4, float2, cl_float)
BASIC_FLOAT_ADDFUNC_TEMPLATE(float4pl, float4, float4, float4, cl_float)
BASIC_FLOAT_ADDFUNC_TEMPLATE(float48pl,float8, float4, float8, cl_double)
BASIC_FLOAT_ADDFUNC_TEMPLATE(float82pl,float8, float8, float2, cl_double)
BASIC_FLOAT_ADDFUNC_TEMPLATE(float84pl,float8, float8, float4, cl_double)
BASIC_FLOAT_ADDFUNC_TEMPLATE(float8pl, float8, float8, float8, cl_double)


#undef BASIC_INT_ADDFUNC_TEMPLATE
#undef BASIC_FLOAT_ADDFUNC_TEMPLATE

/*
 * Functions for addition operator on basic data types
 */
#define BASIC_INT_SUBFUNC_TEMPLATE(name,r_type,x_type,y_type)		\
	DEVICE_FUNCTION(pg_##r_type##_t)								\
	pgfn_##name(kern_context *kcxt,									\
				pg_##x_type##_t arg1, pg_##y_type##_t arg2)			\
	{																\
		pg_##r_type##_t	result;										\
																	\
		result.isnull = arg1.isnull | arg2.isnull;					\
		if (!result.isnull)											\
		{															\
			result.value = arg1.value - arg2.value;					\
			if (!SAMESIGN(arg1.value, arg2.value) &&				\
				!SAMESIGN(result.value, arg1.value))				\
			{														\
				result.isnull = true;								\
				STROM_EREPORT(kcxt, ERRCODE_NUMERIC_VALUE_OUT_OF_RANGE,	\
							  "integer out of range");				\
			}														\
		}															\
		return result;												\
	}

#define BASIC_FLOAT_SUBFUNC_TEMPLATE(name,r_type,x_type,y_type,cast_t) \
	DEVICE_FUNCTION(pg_##r_type##_t)								\
	pgfn_##name(kern_context *kcxt,									\
				pg_##x_type##_t arg1, pg_##y_type##_t arg2)         \
    {																\
		pg_##r_type##_t	result;										\
																	\
		result.isnull = arg1.isnull | arg2.isnull;					\
		if (!result.isnull)											\
		{															\
			result.value = (cast_t)arg1.value - (cast_t)arg2.value;	\
			CHECKFLOATVAL(&kcxt->e, result,							\
						  isinf((cast_t)arg1.value) ||				\
						  isinf((cast_t)arg2.value), true);			\
		}                                                           \
		return result;												\
	}
BASIC_INT_SUBFUNC_TEMPLATE(int1mi,  int1, int1, int1)
BASIC_INT_SUBFUNC_TEMPLATE(int12mi, int2, int1, int2)
BASIC_INT_SUBFUNC_TEMPLATE(int14mi, int4, int1, int4)
BASIC_INT_SUBFUNC_TEMPLATE(int18mi, int8, int1, int8)

BASIC_INT_SUBFUNC_TEMPLATE(int21mi, int2, int2, int1)
BASIC_INT_SUBFUNC_TEMPLATE(int2mi,  int2, int2, int2)
BASIC_INT_SUBFUNC_TEMPLATE(int24mi, int4, int2, int4)
BASIC_INT_SUBFUNC_TEMPLATE(int28mi, int8, int2, int8)

BASIC_INT_SUBFUNC_TEMPLATE(int41mi, int4, int4, int1)
BASIC_INT_SUBFUNC_TEMPLATE(int42mi, int4, int4, int2)
BASIC_INT_SUBFUNC_TEMPLATE(int4mi,  int4, int4, int4)
BASIC_INT_SUBFUNC_TEMPLATE(int48mi, int8, int4, int8)

BASIC_INT_SUBFUNC_TEMPLATE(int81mi, int8, int8, int1)
BASIC_INT_SUBFUNC_TEMPLATE(int82mi, int8, int8, int2)
BASIC_INT_SUBFUNC_TEMPLATE(int84mi, int8, int8, int4)
BASIC_INT_SUBFUNC_TEMPLATE(int8mi,  int8, int8, int8)

BASIC_FLOAT_SUBFUNC_TEMPLATE(float2mi,  float4, float2, float2, cl_float)
BASIC_FLOAT_SUBFUNC_TEMPLATE(float24mi, float4, float2, float4, cl_float)
BASIC_FLOAT_SUBFUNC_TEMPLATE(float28mi, float8, float2, float8, cl_double)
BASIC_FLOAT_SUBFUNC_TEMPLATE(float42mi, float4, float4, float2, cl_float)
BASIC_FLOAT_SUBFUNC_TEMPLATE(float4mi,  float4, float4, float4, cl_float)
BASIC_FLOAT_SUBFUNC_TEMPLATE(float48mi, float8, float4, float8, cl_double)
BASIC_FLOAT_SUBFUNC_TEMPLATE(float82mi, float8, float8, float2, cl_double)
BASIC_FLOAT_SUBFUNC_TEMPLATE(float84mi, float8, float8, float4, cl_double)
BASIC_FLOAT_SUBFUNC_TEMPLATE(float8mi,  float8, float8, float8, cl_double)

#undef BASIC_INT_SUBFUNC_TEMPLATE
#undef BASIC_FLOAT_SUBFUNC_TEMPLATE

/*
 * Functions for multiplication operator on basic data types
 */
DEVICE_INLINE(cl_bool)
__mul_s8_overflow(cl_char a, cl_char b, cl_char *p_result)
{
	cl_int		r = (cl_int) a * (cl_int) b;

	if (r > SCHAR_MAX || r < SCHAR_MIN)
		return true;
	*p_result = r;
	return false;
}

DEVICE_INLINE(cl_bool)
__mul_s16_overflow(cl_short a, cl_short b, cl_short *p_result)
{
	cl_int		r = (cl_int) a * (cl_int) b;

	if (r > SHRT_MAX || r < SHRT_MIN)
		return true;
	*p_result = r;
	return false;
}

DEVICE_INLINE(cl_bool)
__mul_s32_overflow(cl_int a, cl_int b, cl_int *p_result)
{
	cl_long		r = (cl_long) a * (cl_long) b;

	if (r > INT_MAX || r < INT_MIN)
		return true;
	*p_result = r;
	return false;
}

DEVICE_INLINE(cl_bool)
__mul_s64_overflow(cl_long a, cl_long b, cl_long *p_result)
{
	cl_long		hi, lo;

	asm volatile("mul.lo.s64 %0, %2, %3;\n"
				 "mul.hi.s64 %1, %2, %3;"
				 : "=l"(lo), "=l"(hi)
				 : "l"(a), "l"(b));
	if (((a ^ b) >> 63) == 0)
	{
		if (hi != 0UL)
		{
			printf("gid=%u a=%ld b=%ld hi=%lx lo=%lx\n", get_global_id(), a, b, hi, lo);
			return true;	/* must be positive */
		}
	}
	else
	{
		if (hi != ~0UL)
		{
			printf("GID=%u a=%ld b=%ld hi=%lx lo=%lx\n", get_global_id(), a, b, hi, lo);
			return true;	/* must be negative */
		}
	}
	*p_result = lo;
	return false;
}

DEVICE_FUNCTION(pg_int1_t)
pgfn_int1mul(kern_context *kcxt, pg_int1_t arg1, pg_int1_t arg2)
{
	pg_int1_t	result;

	result.isnull = arg1.isnull | arg2.isnull;
	if (!result.isnull && __mul_s8_overflow(arg1.value, arg2.value,
											&result.value))
	{
		result.isnull = true;
		STROM_EREPORT(kcxt, ERRCODE_NUMERIC_VALUE_OUT_OF_RANGE,
					  "tinyint out of range");
	}
	return result;
}

DEVICE_FUNCTION(pg_int2_t)
pgfn_int12mul(kern_context *kcxt, pg_int1_t arg1, pg_int2_t arg2)
{
	pg_int2_t	result;

	result.isnull = arg1.isnull | arg2.isnull;
	if (!result.isnull && __mul_s16_overflow(arg1.value, arg2.value,
											 &result.value))
	{
		result.isnull = true;
		STROM_EREPORT(kcxt, ERRCODE_NUMERIC_VALUE_OUT_OF_RANGE,
					  "smallint out of range");
	}
	return result;
}

DEVICE_FUNCTION(pg_int4_t)
pgfn_int14mul(kern_context *kcxt, pg_int1_t arg1, pg_int4_t arg2)
{
	pg_int4_t	result;

	result.isnull = arg1.isnull | arg2.isnull;
	if (!result.isnull && __mul_s32_overflow(arg1.value, arg2.value,
											 &result.value))
	{
		result.isnull = true;
		STROM_EREPORT(kcxt, ERRCODE_NUMERIC_VALUE_OUT_OF_RANGE,
					  "integer out of range");
	}
	return result;
}

DEVICE_FUNCTION(pg_int8_t)
pgfn_int18mul(kern_context *kcxt, pg_int1_t arg1, pg_int8_t arg2)
{
	pg_int8_t	result;

	result.isnull = arg1.isnull | arg2.isnull;
	if (!result.isnull && __mul_s64_overflow(arg1.value, arg2.value,
											 &result.value))
	{
		result.isnull = true;
		STROM_EREPORT(kcxt, ERRCODE_NUMERIC_VALUE_OUT_OF_RANGE,
					  "bigint out of range");
	}
	return result;
}

DEVICE_FUNCTION(pg_int2_t)
pgfn_int21mul(kern_context *kcxt, pg_int2_t arg1, pg_int2_t arg2)
{
	pg_int2_t	result;

	result.isnull = arg1.isnull | arg2.isnull;
	if (!result.isnull && __mul_s16_overflow(arg1.value, arg2.value,
											 &result.value))
	{
		result.isnull = true;
		STROM_EREPORT(kcxt, ERRCODE_NUMERIC_VALUE_OUT_OF_RANGE,
					  "smallint out of range");
	}
	return result;
}

DEVICE_FUNCTION(pg_int2_t)
pgfn_int2mul(kern_context *kcxt, pg_int2_t arg1, pg_int2_t arg2)
{
	pg_int2_t	result;

	result.isnull = arg1.isnull | arg2.isnull;
	if (!result.isnull && __mul_s16_overflow(arg1.value, arg2.value,
											 &result.value))
	{
		result.isnull = true;
		STROM_EREPORT(kcxt, ERRCODE_NUMERIC_VALUE_OUT_OF_RANGE,
					  "smallint out of range");
	}
	return result;
}

DEVICE_FUNCTION(pg_int4_t)
pgfn_int24mul(kern_context *kcxt, pg_int2_t arg1, pg_int4_t arg2)
{
	pg_int4_t	result;

	result.isnull = arg1.isnull | arg2.isnull;
	if (!result.isnull && __mul_s32_overflow(arg1.value, arg2.value,
											 &result.value))
	{
		result.isnull = true;
		STROM_EREPORT(kcxt, ERRCODE_NUMERIC_VALUE_OUT_OF_RANGE,
					  "integer out of range");
	}
	return result;
}

DEVICE_FUNCTION(pg_int8_t)
pgfn_int28mul(kern_context *kcxt, pg_int2_t arg1, pg_int8_t arg2)
{
	pg_int8_t	result;

	result.isnull = arg1.isnull | arg2.isnull;
	if (!result.isnull && __mul_s64_overflow(arg1.value, arg2.value,
											 &result.value))
	{
		result.isnull = true;
		STROM_EREPORT(kcxt, ERRCODE_NUMERIC_VALUE_OUT_OF_RANGE,
					  "bigint out of range");
	}
	return result;
}

DEVICE_FUNCTION(pg_int4_t)
pgfn_int41mul(kern_context *kcxt, pg_int4_t arg1, pg_int2_t arg2)
{
	pg_int4_t	result;

	result.isnull = arg1.isnull | arg2.isnull;
	if (!result.isnull && __mul_s32_overflow(arg1.value, arg2.value,
											 &result.value))
	{
		result.isnull = true;
		STROM_EREPORT(kcxt, ERRCODE_NUMERIC_VALUE_OUT_OF_RANGE,
					  "integer out of range");
	}
	return result;
}

DEVICE_FUNCTION(pg_int4_t)
pgfn_int42mul(kern_context *kcxt, pg_int4_t arg1, pg_int2_t arg2)
{
	pg_int4_t	result;

	result.isnull = arg1.isnull | arg2.isnull;
	if (!result.isnull && __mul_s32_overflow(arg1.value, arg2.value,
											 &result.value))
	{
		result.isnull = true;
		STROM_EREPORT(kcxt, ERRCODE_NUMERIC_VALUE_OUT_OF_RANGE,
					  "integer out of range");
	}
	return result;
}

DEVICE_FUNCTION(pg_int4_t)
pgfn_int4mul(kern_context *kcxt, pg_int4_t arg1, pg_int4_t arg2)
{
	pg_int4_t	result;

	result.isnull = arg1.isnull | arg2.isnull;
	if (!result.isnull && __mul_s32_overflow(arg1.value, arg2.value,
											 &result.value))
	{
		result.isnull = true;
		STROM_EREPORT(kcxt, ERRCODE_NUMERIC_VALUE_OUT_OF_RANGE,
					  "integer out of range");
	}
	return result;
}

DEVICE_FUNCTION(pg_int8_t)
pgfn_int48mul(kern_context *kcxt, pg_int4_t arg1, pg_int8_t arg2)
{
	pg_int8_t	result;

	result.isnull = arg1.isnull | arg2.isnull;
	if (!result.isnull && __mul_s64_overflow(arg1.value, arg2.value,
											 &result.value))
	{
		result.isnull = true;
		STROM_EREPORT(kcxt, ERRCODE_NUMERIC_VALUE_OUT_OF_RANGE,
					  "bigint out of range");
	}
	return result;
}

DEVICE_FUNCTION(pg_int8_t)
pgfn_int82mul(kern_context *kcxt, pg_int8_t arg1, pg_int2_t arg2)
{
	pg_int8_t	result;

	result.isnull = arg1.isnull | arg2.isnull;
	if (!result.isnull && __mul_s64_overflow(arg1.value, arg2.value,
											 &result.value))
	{
		result.isnull = true;
		STROM_EREPORT(kcxt, ERRCODE_NUMERIC_VALUE_OUT_OF_RANGE,
					  "bigint out of range");
	}
	return result;
}

DEVICE_FUNCTION(pg_int8_t)
pgfn_int84mul(kern_context *kcxt, pg_int8_t arg1, pg_int4_t arg2)
{
	pg_int8_t	result;

	result.isnull = arg1.isnull | arg2.isnull;
	if (!result.isnull && __mul_s64_overflow(arg1.value, arg2.value,
											 &result.value))
	{
		result.isnull = true;
		STROM_EREPORT(kcxt, ERRCODE_NUMERIC_VALUE_OUT_OF_RANGE,
					  "bigint out of range");
	}
	return result;
}

DEVICE_FUNCTION(pg_int8_t)
pgfn_int8mul(kern_context *kcxt, pg_int8_t arg1, pg_int8_t arg2)
{
	pg_int8_t	result;

	result.isnull = arg1.isnull | arg2.isnull;
	if (!result.isnull && __mul_s64_overflow(arg1.value, arg2.value,
											 &result.value))
	{
		result.isnull = true;
		STROM_EREPORT(kcxt, ERRCODE_NUMERIC_VALUE_OUT_OF_RANGE,
					  "bigint out of range");
	}
	return result;
}

DEVICE_FUNCTION(pg_float4_t)
pgfn_float2mul(kern_context *kcxt, pg_float2_t arg1, pg_float2_t arg2)
{
	cl_float	value1 = arg1.value;
	cl_float	value2 = arg2.value;
	pg_float4_t	result;

	result.isnull = arg1.isnull | arg2.isnull;
	if (!result.isnull)
	{
		result.value = value1 * value2;
		CHECKFLOATVAL(&kcxt->e, result,
					  isinf(value1) * isinf(value2),
					  value1 == 0.0 || value2 == 0.0);
	}
	return result;
}


DEVICE_FUNCTION(pg_float4_t)
pgfn_float24mul(kern_context *kcxt, pg_float2_t arg1, pg_float4_t arg2)
{
	cl_float	value1 = arg1.value;
	pg_float4_t	result;

	result.isnull = arg1.isnull | arg2.isnull;
	if (!result.isnull)
	{
		result.value = value1 * arg2.value;
		CHECKFLOATVAL(&kcxt->e, result,
					  isinf(value1) || isinf(arg2.value),
					  value1 == 0.0 || arg2.value == 0.0);
	}
	return result;
}

DEVICE_FUNCTION(pg_float8_t)
pgfn_float28mul(kern_context *kcxt, pg_float2_t arg1, pg_float8_t arg2)
{
	cl_double	value1 = arg1.value;
	pg_float8_t	result;

	result.isnull = arg1.isnull | arg2.isnull;
	if (!result.isnull)
	{
		result.value = value1 * arg2.value;
		CHECKFLOATVAL(&kcxt->e, result,
					  isinf(value1) || isinf(arg2.value),
					  value1 == 0.0 || arg2.value == 0.0);
	}
	return result;
}

DEVICE_FUNCTION(pg_float4_t)
pgfn_float42mul(kern_context *kcxt, pg_float4_t arg1, pg_float2_t arg2)
{
	cl_float	value2 = arg2.value;
	pg_float4_t	result;

	result.isnull = arg1.isnull | arg2.isnull;
	if (!result.isnull)
	{
		result.value = arg1.value * (cl_float)arg2.value;
		CHECKFLOATVAL(&kcxt->e, result,
					  isinf(arg1.value) || isinf(value2),
					  arg1.value == 0.0 || value2 == 0.0);
	}
	return result;
}

DEVICE_FUNCTION(pg_float4_t)
pgfn_float4mul(kern_context *kcxt, pg_float4_t arg1, pg_float4_t arg2)
{
	pg_float4_t	result;

	result.isnull = arg1.isnull | arg2.isnull;
	if (!result.isnull)
	{
		result.value = arg1.value * arg2.value;
		CHECKFLOATVAL(&kcxt->e, result,
					  isinf(arg1.value) || isinf(arg2.value),
					  arg1.value == 0.0 || arg2.value == 0.0);
	}
	return result;
}

DEVICE_FUNCTION(pg_float8_t)
pgfn_float48mul(kern_context *kcxt, pg_float4_t arg1, pg_float8_t arg2)
{
	pg_float8_t	result;

	result.isnull = arg1.isnull | arg2.isnull;
	if (!result.isnull)
	{
		result.value = (cl_double)arg1.value * arg2.value;
		CHECKFLOATVAL(&kcxt->e, result,
					  isinf(arg1.value) || isinf(arg2.value),
					  arg1.value == 0.0 || arg2.value == 0.0);
	}
	return result;
}

DEVICE_FUNCTION(pg_float8_t)
pgfn_float82mul(kern_context *kcxt, pg_float8_t arg1, pg_float2_t arg2)
{
	cl_double	value2 = arg2.value;
	pg_float8_t	result;

	result.isnull = arg1.isnull | arg2.isnull;
	if (!result.isnull)
	{
		result.value = arg1.value * value2;
		CHECKFLOATVAL(&kcxt->e, result,
					  isinf(arg1.value) || isinf(value2),
					  arg1.value == 0.0 || value2 == 0.0);
	}
	return result;
}

DEVICE_FUNCTION(pg_float8_t)
pgfn_float84mul(kern_context *kcxt, pg_float8_t arg1, pg_float4_t arg2)
{
	pg_float8_t	result;

	result.isnull = arg1.isnull | arg2.isnull;
	if (!result.isnull)
	{
		result.value = arg1.value * (cl_double)arg2.value;
		CHECKFLOATVAL(&kcxt->e, result,
					  isinf(arg1.value) || isinf(arg2.value),
					  arg1.value == 0.0 || arg2.value == 0.0);
	}
	return result;
}

DEVICE_FUNCTION(pg_float8_t)
pgfn_float8mul(kern_context *kcxt, pg_float8_t arg1, pg_float8_t arg2)
{
	pg_float8_t	result;

	result.isnull = arg1.isnull | arg2.isnull;
	if (!result.isnull)
	{
		result.value = arg1.value * arg2.value;
		CHECKFLOATVAL(&kcxt->e, result,
					  isinf(arg1.value) || isinf(arg2.value),
					  arg1.value == 0.0 || arg2.value == 0.0);
	}
	return result;
}

/*
 * Functions for division operator on basic data types
 */
#define SAMESIGN(a,b)	(((a) < 0) == ((b) < 0))

DEVICE_FUNCTION(pg_int1_t)
pgfn_int1div(kern_context *kcxt, pg_int1_t arg1, pg_int1_t arg2)
{
	pg_int1_t	result;

	result.isnull = arg1.isnull | arg2.isnull;
	if (!result.isnull)
	{
		if (arg2.value == 0)
		{
			result.isnull = true;
			STROM_EREPORT(kcxt, ERRCODE_DIVISION_BY_ZERO,
						  "division by zero");
		}
		else if (arg2.value == -1)
		{
			if (arg1.value != SCHAR_MIN)
				result.value = -arg1.value;
			else
			{
				result.isnull = true;
				STROM_EREPORT(kcxt, ERRCODE_NUMERIC_VALUE_OUT_OF_RANGE,
                              "tinyint out of range");
			}
		}
		else
		{
			result.value = arg1.value / arg2.value;
		}
	}
	return result;
}

DEVICE_FUNCTION(pg_int2_t)
pgfn_int12div(kern_context *kcxt, pg_int1_t arg1, pg_int2_t arg2)
{
	pg_int2_t	result;

	result.isnull = arg1.isnull | arg2.isnull;
	if (!result.isnull)
	{
		if (arg2.value == 0)
		{
			result.isnull = true;
			STROM_EREPORT(kcxt, ERRCODE_DIVISION_BY_ZERO,
						  "division by zero");
		}
		/* no overflow is possible */
		result.value = arg1.value / arg2.value;
	}
	return result;
}

DEVICE_FUNCTION(pg_int4_t)
pgfn_int14div(kern_context *kcxt, pg_int1_t arg1, pg_int4_t arg2)
{
	pg_int4_t	result;

	result.isnull = arg1.isnull | arg2.isnull;
	if (!result.isnull)
	{
		if (arg2.value == 0)
		{
			result.isnull = true;
			STROM_EREPORT(kcxt, ERRCODE_DIVISION_BY_ZERO,
						  "division by zero");
		}
		/* no overflow is possible */
		result.value = arg1.value / arg2.value;
	}
	return result;
}

DEVICE_FUNCTION(pg_int8_t)
pgfn_int18div(kern_context *kcxt, pg_int1_t arg1, pg_int8_t arg2)
{
	pg_int8_t	result;

	result.isnull = arg1.isnull | arg2.isnull;
	if (!result.isnull)
	{
		if (arg2.value == 0)
		{
			result.isnull = true;
			STROM_EREPORT(kcxt, ERRCODE_DIVISION_BY_ZERO,
						  "division by zero");
		}
		/* no overflow is possible */
		result.value = arg1.value / arg2.value;
	}
	return result;
}

DEVICE_FUNCTION(pg_int2_t)
pgfn_int21div(kern_context *kcxt, pg_int2_t arg1, pg_int1_t arg2)
{
	pg_int2_t	result;

	result.isnull = arg1.isnull | arg2.isnull;
    if (!result.isnull)
	{
		if (arg2.value == 0)
		{
			result.isnull = true;
			STROM_EREPORT(kcxt, ERRCODE_DIVISION_BY_ZERO,
						  "division by zero");
		}
		else if (arg2.value == -1)
		{
			if (arg1.value != SHRT_MIN)
				result.value = -arg1.value;
			else
			{
				result.isnull = true;
				STROM_EREPORT(kcxt, ERRCODE_NUMERIC_VALUE_OUT_OF_RANGE,
							  "smallint out of range");
			}
		}
		else
			result.value = arg1.value / arg2.value;
	}
	return result;
}

DEVICE_FUNCTION(pg_int2_t)
pgfn_int2div(kern_context *kcxt, pg_int2_t arg1, pg_int2_t arg2)
{
	pg_int2_t	result;

	result.isnull = arg1.isnull | arg2.isnull;
    if (!result.isnull)
	{
		if (arg2.value == 0)
		{
			result.isnull = true;
			STROM_EREPORT(kcxt, ERRCODE_DIVISION_BY_ZERO,
						  "division by zero");
		}
		else if (arg2.value == -1)
		{
			if (arg1.value != SHRT_MIN)
				result.value = -arg1.value;
			else
			{
				result.isnull = true;
				STROM_EREPORT(kcxt, ERRCODE_NUMERIC_VALUE_OUT_OF_RANGE,
							  "smallint out of range");
			}
		}
		else
			result.value = arg1.value / arg2.value;
	}
	return result;
}

DEVICE_FUNCTION(pg_int4_t)
pgfn_int24div(kern_context *kcxt, pg_int2_t arg1, pg_int4_t arg2)
{
	pg_int4_t	result;

	result.isnull = arg1.isnull | arg2.isnull;
    if (!result.isnull)
	{
		if (arg2.value == 0)
		{
			result.isnull = true;
			STROM_EREPORT(kcxt, ERRCODE_DIVISION_BY_ZERO,
						  "division by zero");
		}
		else
			result.value = (cl_int) arg1.value / arg2.value;
	}
	return result;
}

DEVICE_FUNCTION(pg_int8_t)
pgfn_int28div(kern_context *kcxt, pg_int2_t arg1, pg_int8_t arg2)
{
	pg_int8_t	result;

	result.isnull = arg1.isnull | arg2.isnull;
    if (!result.isnull)
	{
		if (arg2.value == 0)
		{
			result.isnull = true;
			STROM_EREPORT(kcxt, ERRCODE_DIVISION_BY_ZERO,
                          "division by zero");
		}
		else
			result.value = (cl_long) arg1.value / arg2.value;
	}
	return result;
}

DEVICE_FUNCTION(pg_int4_t)
pgfn_int41div(kern_context *kcxt, pg_int4_t arg1, pg_int1_t arg2)
{
	pg_int4_t	result;

	result.isnull = arg1.isnull | arg2.isnull;
    if (!result.isnull)
    {
		if (arg2.value == 0)
		{
			result.isnull = true;
			STROM_EREPORT(kcxt, ERRCODE_DIVISION_BY_ZERO,
						  "division by zero");
		}
		else if (arg2.value == -1)
		{
			if (arg1.value != INT_MIN)
				result.value = -arg1.value;
			else
			{
				result.isnull = true;
				STROM_EREPORT(kcxt, ERRCODE_NUMERIC_VALUE_OUT_OF_RANGE,
							  "integer out of range");
			}
		}
		else
			result.value = arg1.value / arg2.value;
	}
	return result;
}

DEVICE_FUNCTION(pg_int4_t)
pgfn_int42div(kern_context *kcxt, pg_int4_t arg1, pg_int2_t arg2)
{
	pg_int4_t	result;

	result.isnull = arg1.isnull | arg2.isnull;
    if (!result.isnull)
    {
		if (arg2.value == 0)
		{
			result.isnull = true;
			STROM_EREPORT(kcxt, ERRCODE_DIVISION_BY_ZERO,
						  "division by zero");
		}
		else if (arg2.value == -1)
		{
			if (arg1.value != INT_MIN)
				result.value = -arg1.value;
			else
			{
				result.isnull = true;
				STROM_EREPORT(kcxt, ERRCODE_NUMERIC_VALUE_OUT_OF_RANGE,
							  "integer out of range");
			}
		}
		else
			result.value = arg1.value / arg2.value;
	}
	return result;
}

DEVICE_FUNCTION(pg_int4_t)
pgfn_int4div(kern_context *kcxt, pg_int4_t arg1, pg_int4_t arg2)
{
	pg_int4_t	result;

	result.isnull = arg1.isnull | arg2.isnull;
    if (!result.isnull)
	{
		if (arg2.value == 0)
		{
			result.isnull = true;
			STROM_EREPORT(kcxt, ERRCODE_DIVISION_BY_ZERO,
						  "division by zero");
		}
		else if (arg2.value == -1)
		{
			if (arg1.value != INT_MIN)
				result.value = -arg1.value;
			else
			{
				result.isnull = true;
				STROM_EREPORT(kcxt, ERRCODE_NUMERIC_VALUE_OUT_OF_RANGE,
							  "integer out of range");
			}
		}
		else
			result.value = arg1.value / arg2.value;
	}
	return result;
}

DEVICE_FUNCTION(pg_int8_t)
pgfn_int48div(kern_context *kcxt, pg_int4_t arg1, pg_int8_t arg2)
{
	pg_int8_t	result;

	result.isnull = arg1.isnull | arg2.isnull;
    if (!result.isnull)
	{
		if (arg2.value == 0)
		{
			result.isnull = true;
			STROM_EREPORT(kcxt, ERRCODE_DIVISION_BY_ZERO,
                          "division by zero");
		}
		else
			result.value = (cl_long) arg1.value / arg2.value;
	}
	return result;
}

DEVICE_FUNCTION(pg_int8_t)
pgfn_int81div(kern_context *kcxt, pg_int8_t arg1, pg_int1_t arg2)
{
	pg_int8_t	result;

	result.isnull = arg1.isnull | arg2.isnull;
	if (!result.isnull)
	{
		if (arg2.value == 0)
		{
			result.isnull = true;
			STROM_EREPORT(kcxt, ERRCODE_DIVISION_BY_ZERO,
						  "division by zero");
		}
		else if (arg2.value == -1)
		{
			if (arg1.value != LONG_MIN)
				result.value = -arg1.value;
			else
			{
				result.isnull = true;
				STROM_EREPORT(kcxt, ERRCODE_NUMERIC_VALUE_OUT_OF_RANGE,
							  "bigint out of range");
			}
		}
		else
			result.value = arg1.value / arg2.value;
	}
	return result;
}

DEVICE_FUNCTION(pg_int8_t)
pgfn_int82div(kern_context *kcxt, pg_int8_t arg1, pg_int2_t arg2)
{
	pg_int8_t	result;

	result.isnull = arg1.isnull | arg2.isnull;
	if (!result.isnull)
	{
		if (arg2.value == 0)
		{
			result.isnull = true;
			STROM_EREPORT(kcxt, ERRCODE_DIVISION_BY_ZERO,
						  "division by zero");
		}
		else if (arg2.value == -1)
		{
			if (arg1.value != LONG_MIN)
				result.value = -arg1.value;
			else
			{
				result.isnull = true;
				STROM_EREPORT(kcxt, ERRCODE_NUMERIC_VALUE_OUT_OF_RANGE,
							  "bigint out of range");
			}
		}
		else
			result.value = arg1.value / arg2.value;
	}
	return result;
}

DEVICE_FUNCTION(pg_int8_t)
pgfn_int84div(kern_context *kcxt, pg_int8_t arg1, pg_int4_t arg2)
{
	pg_int8_t	result;

	result.isnull = arg1.isnull | arg2.isnull;
	if (!result.isnull)
	{
		if (arg2.value == 0)
		{
			result.isnull = true;
			STROM_EREPORT(kcxt, ERRCODE_DIVISION_BY_ZERO,
						  "division by zero");
		}
		else if (arg2.value == -1)
		{
			if (arg1.value != LONG_MIN)
				result.value = -arg1.value;
			else
			{
				result.isnull = true;
				STROM_EREPORT(kcxt, ERRCODE_NUMERIC_VALUE_OUT_OF_RANGE,
							  "bigint out of range");
			}
		}
		else
			result.value = arg1.value / arg2.value;
	}
	return result;
}

DEVICE_FUNCTION(pg_int8_t)
pgfn_int8div(kern_context *kcxt, pg_int8_t arg1, pg_int8_t arg2)
{
	pg_int8_t	result;

	result.isnull = arg1.isnull | arg2.isnull;
	if (!result.isnull)
	{
		if (arg2.value == 0)
		{
			result.isnull = true;
			STROM_EREPORT(kcxt, ERRCODE_DIVISION_BY_ZERO,
						  "division by zero");
		}
		else if (arg2.value == -1)
		{
			if (arg1.value != LONG_MIN)
				result.value = -arg1.value;
			else
			{
				result.isnull = true;
				STROM_EREPORT(kcxt, ERRCODE_NUMERIC_VALUE_OUT_OF_RANGE,
							  "bigint out of range");
			}
		}
		else
			result.value = arg1.value / arg2.value;
	}
	return result;
}

DEVICE_FUNCTION(pg_float4_t)
pgfn_float2div(kern_context *kcxt, pg_float2_t arg1, pg_float2_t arg2)
{
	cl_float	value1 = arg1.value;
	cl_float	value2 = arg2.value;
	pg_float4_t	result;

	result.isnull = arg1.isnull | arg2.isnull;
    if (!result.isnull)
    {
		if (value2 == 0.0)
		{
			result.isnull = true;
			STROM_EREPORT(kcxt, ERRCODE_DIVISION_BY_ZERO,
						  "division by zero");
		}
		else
		{
			result.value = value1 / value2;
			CHECKFLOATVAL(&kcxt->e, result,
						  isinf(value1) || isinf(value2),
						  value1 == 0.0);
		}
	}
	return result;
}


DEVICE_FUNCTION(pg_float4_t)
pgfn_float24div(kern_context *kcxt, pg_float2_t arg1, pg_float4_t arg2)
{
	cl_float	value1 = arg1.value;
	pg_float4_t	result;

	result.isnull = arg1.isnull | arg2.isnull;
    if (!result.isnull)
    {
		if (arg2.value == 0.0)
		{
			result.isnull = true;
			STROM_EREPORT(kcxt, ERRCODE_DIVISION_BY_ZERO,
						  "division by zero");
		}
		else
		{
			result.value = value1 / arg2.value;
			CHECKFLOATVAL(&kcxt->e, result,
						  isinf(value1) || isinf(arg2.value),
						  value1 == 0.0);
		}
	}
	return result;
}

DEVICE_FUNCTION(pg_float8_t)
pgfn_float28div(kern_context *kcxt, pg_float2_t arg1, pg_float8_t arg2)
{
	cl_double	value1 = arg1.value;
	pg_float8_t	result;

	result.isnull = arg1.isnull | arg2.isnull;
    if (!result.isnull)
    {
		if (arg2.value == 0.0)
		{
			result.isnull = true;
			STROM_EREPORT(kcxt, ERRCODE_DIVISION_BY_ZERO,
						  "division by zero");
		}
		else
		{
			result.value = value1 / arg2.value;
			CHECKFLOATVAL(&kcxt->e, result,
						  isinf(value1) || isinf(arg2.value),
						  value1 == 0.0);
		}
	}
	return result;
}

DEVICE_FUNCTION(pg_float4_t)
pgfn_float42div(kern_context *kcxt, pg_float4_t arg1, pg_float2_t arg2)
{
	cl_float	value2 = arg2.value;
	pg_float4_t	result;

	result.isnull = arg1.isnull | arg2.isnull;
    if (!result.isnull)
    {
		if (value2 == 0.0)
		{
			result.isnull = true;
			STROM_EREPORT(kcxt, ERRCODE_DIVISION_BY_ZERO,
						  "division by zero");
		}
		else
		{
			result.value = arg1.value / value2;
			CHECKFLOATVAL(&kcxt->e, result,
						  isinf(arg1.value) || isinf(value2),
						  arg1.value == 0.0);
		}
	}
	return result;
}

DEVICE_FUNCTION(pg_float4_t)
pgfn_float4div(kern_context *kcxt, pg_float4_t arg1, pg_float4_t arg2)
{
	pg_float4_t	result;

	result.isnull = arg1.isnull | arg2.isnull;
    if (!result.isnull)
    {
		if (arg2.value == 0.0)
		{
			result.isnull = true;
			STROM_EREPORT(kcxt, ERRCODE_DIVISION_BY_ZERO,
						  "division by zero");
		}
		else
		{
			result.value = arg1.value / arg2.value;
			CHECKFLOATVAL(&kcxt->e, result,
						  isinf(arg1.value) || isinf(arg2.value),
						  arg1.value == 0.0);
		}
	}
	return result;
}

DEVICE_FUNCTION(pg_float8_t)
pgfn_float48div(kern_context *kcxt, pg_float4_t arg1, pg_float8_t arg2)
{
	pg_float8_t	result;

	result.isnull = arg1.isnull | arg2.isnull;
    if (!result.isnull)
    {
		if (arg2.value == 0.0)
		{
			result.isnull = true;
			STROM_EREPORT(kcxt, ERRCODE_DIVISION_BY_ZERO,
						  "division by zero");
		}
		else
		{
			result.value = (cl_double)arg1.value / arg2.value;
			CHECKFLOATVAL(&kcxt->e, result,
						  isinf(arg1.value) || isinf(arg2.value),
						  arg1.value == 0.0);
		}
	}
	return result;
}

DEVICE_FUNCTION(pg_float8_t)
pgfn_float82div(kern_context *kcxt, pg_float8_t arg1, pg_float2_t arg2)
{
	cl_double	value2 = arg2.value;
	pg_float8_t	result;

	result.isnull = arg1.isnull | arg2.isnull;
    if (!result.isnull)
    {
		if (value2 == 0.0)
		{
			result.isnull = true;
			STROM_EREPORT(kcxt, ERRCODE_DIVISION_BY_ZERO,
						  "division by zero");
		}
		else
		{
			result.value = arg1.value / value2;
			CHECKFLOATVAL(&kcxt->e, result,
						  isinf(arg1.value) || isinf(value2),
						  arg1.value == 0.0);
		}
	}
	return result;
}

DEVICE_FUNCTION(pg_float8_t)
pgfn_float84div(kern_context *kcxt, pg_float8_t arg1, pg_float4_t arg2)
{
	pg_float8_t	result;

	result.isnull = arg1.isnull | arg2.isnull;
    if (!result.isnull)
    {
		if (arg2.value == 0.0)
		{
			result.isnull = true;
			STROM_EREPORT(kcxt, ERRCODE_DIVISION_BY_ZERO,
						  "division by zero");
		}
		else
		{
			result.value = arg1.value / (cl_double)arg2.value;
			CHECKFLOATVAL(&kcxt->e, result,
						  isinf(arg1.value) || isinf(arg2.value),
						  arg1.value == 0.0);
		}
	}
	return result;
}

DEVICE_FUNCTION(pg_float8_t)
pgfn_float8div(kern_context *kcxt, pg_float8_t arg1, pg_float8_t arg2)
{
	pg_float8_t	result;

	result.isnull = arg1.isnull | arg2.isnull;
    if (!result.isnull)
    {
		if (arg2.value == 0.0)
		{
			result.isnull = true;
			STROM_EREPORT(kcxt, ERRCODE_DIVISION_BY_ZERO,
						  "division by zero");
		}
		else
		{
			result.value = arg1.value / arg2.value;
			CHECKFLOATVAL(&kcxt->e, result,
						  isinf(arg1.value) || isinf(arg2.value),
						  arg1.value == 0.0);
		}
	}
	return result;
}

/*
 * Functions for modulo operator on basic data types
 */
#define BASIC_INT_MODFUNC_TEMPLATE(name,d_type)						\
	DEVICE_FUNCTION(pg_##d_type##_t)								\
	pgfn_##name(kern_context *kcxt,									\
				pg_##d_type##_t arg1, pg_##d_type##_t arg2)			\
	{																\
		pg_##d_type##_t	result;										\
																	\
		result.isnull = arg1.isnull | arg2.isnull;					\
		if (!result.isnull)											\
		{															\
			if (arg2.value == 0)									\
			{														\
				result.isnull = true;								\
				STROM_EREPORT(kcxt, ERRCODE_DIVISION_BY_ZERO,		\
							  "division by zero");					\
			}														\
			else if (arg2.value == -1)								\
				result.value = 0;									\
			else													\
				result.value = arg1.value % arg2.value;				\
		}															\
		return result;												\
	}

BASIC_INT_MODFUNC_TEMPLATE(int2mod, int2)
BASIC_INT_MODFUNC_TEMPLATE(int4mod, int4)
BASIC_INT_MODFUNC_TEMPLATE(int8mod, int8)

#undef BASIC_INT_MODFUNC_TEMPLATE
